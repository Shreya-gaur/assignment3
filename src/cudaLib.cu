#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

#define BLOCK_SIZE 16
__device__ __constant__ float bias_d[512];

void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	std::cout << "Lazy, you are!\n";
	std::cout << "Write code, you must\n";

	return 0;
}

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 3.14159f;

	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}




int runGpuMedianFilter (std::string imgPath, std::string outPath, MedianFilterArgs args) {
	
	std::cout << "Lazy, you are! ... ";
	std::cout << "Filter pixels, you must! ... ";

	return 0;
}

int medianFilter_gpu (uint8_t inPixels, ImageDim imgDim, 
	uint8_t outPixels, MedianFilterArgs args) {

	return 0;
}


int runGpuConv (int argc, char ** argv) {

	TensorShape iShape = AlexL1_InShape;
	TensorShape fShape = AlexL1_FilterShape;
	ConvLayerArgs convArgs = AlexL1_ConvArgs;

	std::cout << "Evaluate convolution : \n";
	std::cout << "Input : " << iShape << " \n";
	std::cout << "Filter : " << fShape << " \n";

	TensorShape oShape;

	uint64_t errorCount = evaluateGpuConv(iShape, fShape, oShape, convArgs);
	std::cout << "\nFound " << errorCount << " / " << tensorSize(oShape) << " errors \n";
	return 0;
}

uint64_t evaluateGpuConv (TensorShape iShape, TensorShape fShape, 
	TensorShape & oShape, ConvLayerArgs args) {

	uint64_t errorCount = 0;

	float *input_d, *output_d, *filter_d;

	if (iShape.channels == 0) iShape.channels = 1;

	printf("iShape.height: %i, iShape.width: %i, iShape.channel: %i, iShape.count: %i \n", iShape.height, iShape.width, iShape.channels, iShape.count);
	printf("args.strideH: %i, args.strideW: %i, args.padH: %i, args.padW: %i \n", args.strideH, args.strideW, args.padH, args.padW);
	printf("filter.height: %i, filter.width: %i, filter.channel: %i, filter.batch: %i \n", fShape.height, fShape.width, fShape.channels, fShape.count); 

	oShape.height 	= (iShape.height + 2 * args.padH - fShape.height) / args.strideH + 1;
	oShape.width	= (iShape.width  + 2 * args.padW - fShape.width)  / args.strideW + 1;
	oShape.channels	= (fShape.count);
	oShape.count 	= (iShape.count);				//	Might scale to batch size

	printf("oShape.height: %i, oShape.width: %i, oShape.channel: %i \n", oShape.height, oShape.width, oShape.channels);

	float * in = nullptr;
	float * filter = nullptr;
	float * bias = nullptr; 
	float * out = nullptr;
	float * out_cpu = nullptr;

	int retVal;
	retVal = makeTensor(&in, iShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n" ;
		return -1;
	}
	retVal = makeTensor(&filter, fShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n" ;
		return -1;
	}
	retVal = makeVector(&bias, oShape.channels);
	if (retVal != 0) {
		std::cout << "Unable to make vector \n" ;
		return -1;
	}

	if(args.padH != 0 || args.padW != 0){

		TensorShape_t padShape;

		padShape.count = iShape.count;
		padShape.channels = iShape.channels;
		padShape.height = iShape.height + 2 * args.padH;
		padShape.width = iShape.width + 2 * args.padW;

		float* paddedin = (float*) malloc(tensorSize(padShape) *  sizeof(float)); 
		
		for(int ch =0; ch< padShape.channels; ch++){
			for( int i = 0; i < padShape.height; i++ ) {
				for( int j = 0; j < padShape.width; j++ ) {
					int paddedPixelPos = ch * padShape.height * padShape.width + i * padShape.width + j;

					if( i >= args.padH && i < iShape.height + args.padH &&
						j >= args.padW && j < iShape.width + args.padW ) {
						int pixelPos = ch * iShape.height * iShape.width + ( i - args.padH ) * iShape.width + ( j - args.padW);
						paddedin[paddedPixelPos] = in[pixelPos];
					} else {
						paddedin[paddedPixelPos] = 0.0;
					}
				}
			}
		}
		
		free(in);

		in = paddedin;
		iShape.width = padShape.width;
		iShape.height = padShape.height;

	}

	if(hipMalloc(&input_d,  tensorSize(iShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(iShape) * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! INPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	if(hipMalloc(&filter_d, tensorSize(fShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(fShape) * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! FILTER MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	// if(hipMalloc(&bias_d,  oShape.channels * sizeof(float))!=hipSuccess){
	// 	std::cout<< "Size Requested: "<< oShape.channels * sizeof(float);
	// 	std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! BIAS MEMORY ALLOCATION FAILURE \n";
	// 	return -1;
	// }

	std::cout << "OutShape : " << oShape << " \n";
	out = (float *) malloc (tensorSize(oShape) * sizeof(float));

	if(hipMalloc(&output_d,tensorSize(oShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(oShape) * sizeof(float);
		std::cout<< "ERROR ERROR!!!!! RUN FOR THE HILLS!!!!!OUTPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	// std::cout << "Input" << "\n"; 

	// for(uint32_t n = 0; n < iShape.count ; n++){
	// 	std::cout<< "Batch: "<< n << "\n";
	// 	for(uint32_t ch = 0; ch < iShape.channels; ch++){
	// 		std::cout<< "Channel: "<< ch << "\n";
	// 		for (uint32_t i = 0; i < iShape.height; i++){
	// 			for(uint32_t j = 0; j < iShape.width; j++){
	// 				std::cout << in[ch * iShape.width * iShape.height + i * iShape.width + j] << " ";
	// 			}
	// 			std::cout << "\n";
	// 		}
	// 		std::cout << "\n";
	// 	}
	// }


	// std::cout << "\n"; 	

	// std::cout << "Filter" << "\n"; 

	// for(uint32_t ch = 0; ch < fShape.channels; ch++){
	// 	std::cout<< "Channel: "<< ch << "\n";
	// 	for (uint32_t i = 0; i < fShape.height; i++){
	// 		for(uint32_t j = 0; j < fShape.width; j++){
	// 			// std::cout << output[ch * oShape.width * oShape.height + i * oShape.width + j] << " @ (" << i << ", " << j << ")" << "\n";
	// 			std::cout << filter[ch * fShape.width * fShape.height + i * fShape.width + j] << " ";
	// 		}
	// 		std::cout << "\n";
	// 	}
	// 	std::cout << "\n";
	// }

	// std::cout << "Bias" << "\n";  

	// for(uint32_t ch = 0; ch < oShape.channels; ch++ ){
	// 	std::cout<< "Channel: "<< ch << "\n";
	// 	std::cout << bias[ch] << "\n ";
	// }

	// std::cout <<"\n";

	hipMemcpy(input_d, in, tensorSize(iShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(filter_d, filter, tensorSize(fShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(bias_d), bias, oShape.channels * sizeof(float), 0, hipMemcpyHostToDevice);

	int tileW = BLOCK_SIZE + (fShape.width -  args.strideW);
	int tileH = BLOCK_SIZE + (fShape.height - args.strideH);
	int threadPerBlockH = 16;

	int sharedmemSize = tileW * tileH * sizeof(float);

	dim3 dimBlock(tileW, threadPerBlockH, 1);
	printf("\ndimBlock: (%i, %i, %i)\n", tileW, threadPerBlockH, dimBlock.z);
    dim3 dimGrid(ceil((float)iShape.width / (float)BLOCK_SIZE), ceil((float)iShape.height / (float)BLOCK_SIZE), iShape.channels);
	std::cout << "dimGrid: ("<< dimGrid.x << "," << dimGrid.y << "," << dimGrid.z << ")\n";

	convLayer_gpu<<<dimGrid, dimBlock, sharedmemSize>>>(input_d, iShape, filter_d, fShape, bias_d, output_d, oShape, args, iShape.count);


	dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2(ceil((float)oShape.width / (float)BLOCK_SIZE), ceil((float)oShape.height / (float)BLOCK_SIZE), oShape.channels);
	
	bias_Add<<<dimGrid2, dimBlock2>>>(bias_d, output_d, oShape);

	hipMemcpy(out, output_d, tensorSize(oShape) * sizeof(float), hipMemcpyDeviceToHost);	

	// std::cout << "\n"; 	
	// std::cout << "Output GPU" << "\n"; 
	// for(uint32_t n = 0; n < oShape.count; n++){
	// 	std::cout<< "Batch: "<< n << "\n";
	// 	for(uint32_t ch = 0; ch < oShape.channels; ch++){
	// 		std::cout<< "Channel: "<< ch << "\n";
	// 		for (uint32_t i = 0; i < oShape.height; i++){
	// 			for(uint32_t j = 0; j < oShape.width; j++){
	// 				std::cout << out[ch * oShape.width * oShape.height + i * oShape.width + j] << " ";
	// 			}
	// 			std::cout << "\n";
	// 		}
	// 		std::cout << "\n";
	// 	}
	// }

	#ifndef CONV_CHECK_DISABLE
		//	STUDENT: Verify number of errors in output matrix generated by convLayer_gpu
		//	STUDENT: Compare results with CPU output
		//	STUDENT: Return error count
		out_cpu = (float *) malloc (tensorSize(oShape) * sizeof(float));
		
		auto tStart = std::chrono::high_resolution_clock::now();		

		convLayer_cpu(in, iShape, filter, fShape, bias, out_cpu, oShape, args, iShape.count);
		
		auto tEnd= std::chrono::high_resolution_clock::now();

		std::chrono::duration<double> time_span = (tEnd- tStart);

		for(uint32_t ch = 0; ch < oShape.channels; ch++){
			for (uint32_t i = 0; i < oShape.height; i++){
				for(uint32_t j = 0; j < oShape.width; j++){
					float output_gpu = out[ch * oShape.width * oShape.height + i * oShape.width + j];
					float output_cpu = out_cpu[ch * oShape.width * oShape.height + i * oShape.width + j];
					if(floor(fabs(output_gpu - output_cpu)) != 0 ){
						// printf("Error at (%i, %i, %i) -> Actual Value: %f GPU Value: %f\n", i, j, ch, out_cpu[ch * oShape.width * oShape.height + i * oShape.width + j], out[ch * oShape.width * oShape.height + i * oShape.width + j]);
						// printf("Error at (%i, %i, %i) -> Difference: %f\n", i, j, ch, floor(output_gpu - output_cpu));
						errorCount += 1;
					}
				}
			}
		}

		// std::cout << "\n"; 	
		// std::cout << "Output CPU" << "\n"; 
	
		// for(uint32_t n = 0; n < oShape.count ; n++){
		// 	std::cout<< "Batch: "<< n << "\n";
		// 	for(uint32_t ch = 0; ch < oShape.channels; ch++){
		// 		std::cout<< "Channel: "<< ch << "\n";
		// 		for (uint32_t i = 0; i < oShape.height; i++){
		// 			for(uint32_t j = 0; j < oShape.width; j++){
		// 				std::cout << out_cpu[ch * oShape.width * oShape.height + i * oShape.width + j] << " ";
		// 			}
		// 			std::cout << "\n";
		// 		}
		// 		std::cout << "\n";
		// 	}
		// }

		std::cout << "\n"; 
		std::cout << "It took " << time_span.count() << " seconds on CPU.";	

	#endif

	free(in);
	free(filter);
	free(bias);
	free(out);

	hipFree(input_d);
	hipFree(filter_d);
	// hipFree(bias_d);
	hipFree(output_d);

	return errorCount;
}

__global__
void bias_Add (float * bias, float * output, TensorShape oShape){
	uint32_t row = blockDim.y * blockIdx.y + threadIdx.y;
	uint32_t col = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t channel = blockIdx.z;

	if(row < oShape.height && col < oShape.width && channel < oShape.channels){
		output[channel * oShape.width * oShape.height + row * oShape.width + col] += bias_d[channel];
	}

	return;
}


__global__
void convLayer_gpu ( float * input, TensorShape iShape, 
	float * filter, TensorShape fShape, 
	float * bias, float * output, TensorShape oShape, //removed & after TensorShap 
	ConvLayerArgs args, uint32_t batchSize) {

    extern __shared__ float shrinput[];

	const uint32_t tileW = BLOCK_SIZE + (fShape.width - args.strideW);
	const uint32_t tileH = BLOCK_SIZE + (fShape.height - args.strideH);
	
	const uint32_t nosubBlk = ceil((float)tileH / (float)blockDim.y);

    const uint32_t blockStartCol = blockIdx.x * BLOCK_SIZE;
    const uint32_t blockEndCol = blockStartCol + BLOCK_SIZE;
    const uint32_t blockStartRow = blockIdx.y * BLOCK_SIZE;
    const uint32_t blockEndRow = blockStartRow + BLOCK_SIZE;

    const uint32_t tileStartCol = blockStartCol;
    const uint32_t tileEndCol = blockEndCol + (fShape.width - args.strideW);
    const uint32_t tileEndClampedCol = min(tileEndCol, iShape.width);

    const uint32_t tileStartRow = blockStartRow;
    const uint32_t tileEndRow = blockEndRow + fShape.width - args.strideW;
    const uint32_t tileEndClampedRow = min(tileEndRow, iShape.height);

    uint32_t tilePosCol = threadIdx.x;
    uint32_t iPosCol = tileStartCol + tilePosCol;

	for( uint32_t subBlockNo = 0; subBlockNo < nosubBlk; subBlockNo++ ) {
		uint32_t tilePosRow = subBlockNo * blockDim.y + threadIdx.y;		
		uint32_t iPosRow = tileStartRow + tilePosRow;	
		uint32_t tilePos = tilePosRow * tileW + tilePosCol;

		if( iPosCol < tileEndClampedCol && iPosRow < tileEndClampedRow ) {
			uint32_t iPos = blockIdx.z * iShape.width * iShape.height + iPosRow * iShape.width + iPosCol;
			shrinput[tilePos] = input[iPos];
		}
	
	}

	__syncthreads();

    tilePosCol = threadIdx.x;
    iPosCol = tileStartCol + tilePosCol * args.strideW;

	for(uint32_t m = 0; m < oShape.channels; m++){

		float conv_op = 0;
		
		for(uint32_t subBlockNo = 0; subBlockNo < nosubBlk; subBlockNo++ ) {

			uint32_t tilePosRow = subBlockNo * blockDim.y + threadIdx.y;
			uint32_t iPosRow = tileStartRow + tilePosRow * args.strideH;

			if( iPosCol >= tileStartCol && iPosCol < tileEndClampedCol - (fShape.width - args.strideW) &&
				iPosRow >= tileStartRow && iPosRow < tileEndClampedRow - (fShape.height - args.strideH) ) {
				
				uint32_t oPixelPosCol = iPosCol / args.strideW;
				uint32_t oPixelPosRow = iPosRow / args.strideH;
				uint32_t oPixelPos = oPixelPosRow * oShape.width + oPixelPosCol;
				uint32_t tilePos = (tilePosRow * args.strideH) * tileW + (tilePosCol * args.strideW);

				for( uint32_t i = 0; i < fShape.height; i++ ) {
					for( uint32_t j = 0; j < fShape.width; j++ ) {
							int tilePosOffset = i * tileW + j;
							int coefPos = m * fShape.channels * fShape.width * fShape.height + blockIdx.z * fShape.width * fShape.height + i * fShape.width + j;
							int input_idx = tilePos + tilePosOffset;
							conv_op += shrinput[input_idx] * filter[coefPos];
					}
				}

				__syncthreads();

				atomicAdd(&output[m * oShape.height * oShape.width + oPixelPos], conv_op);

			}

		}
	}
	return;
}


int runGpuGemm (int argc, char ** argv) {

	TensorShape aShape = {1, 1, 6, 4};
	TensorShape bShape = {1, 1, 4, 8};
	TensorShape cShape;
	GemmLayerArgs args = {2, 2, 1};

	// evaluateGpuGemm();

	return 0;
}

int evaluateGpuGemm() {

	return 0;
}

//	STUDENT: Add functions here

