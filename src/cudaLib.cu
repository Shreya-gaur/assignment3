#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

#define BLOCK_SIZE 8


void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	std::cout << "Lazy, you are!\n";
	std::cout << "Write code, you must\n";

	return 0;
}

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 3.14159f;

	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}




int runGpuMedianFilter (std::string imgPath, std::string outPath, MedianFilterArgs args) {
	
	std::cout << "Lazy, you are! ... ";
	std::cout << "Filter pixels, you must! ... ";

	return 0;
}

int medianFilter_gpu (uint8_t inPixels, ImageDim imgDim, 
	uint8_t outPixels, MedianFilterArgs args) {

	return 0;
}


int runGpuConv (int argc, char ** argv) {

	TensorShape iShape = AlexL1_InShape;
	TensorShape fShape = AlexL1_FilterShape;
	ConvLayerArgs convArgs = AlexL1_ConvArgs;

	std::cout << "Evaluate convolution : \n";
	std::cout << "Input : " << iShape << " \n";
	std::cout << "Filter : " << fShape << " \n";

	TensorShape oShape;

	uint64_t errorCount = evaluateGpuConv(iShape, fShape, oShape, convArgs);
	std::cout << "\nFound " << errorCount << " / " << tensorSize(oShape) << " errors \n";
	return 0;
}

uint64_t evaluateGpuConv (TensorShape iShape, TensorShape fShape, 
	TensorShape & oShape, ConvLayerArgs args) {

	uint64_t errorCount = 0;

	//	STUDENT: Add code here --> Added

	float *input_d, *output_d, *bias_d, *filter_d;

	if (iShape.channels == 0) iShape.channels = 1;

	printf("iShape.height: %i, iShape.width: %i, iShape.channel: %i, iShape.count: %i \n", iShape.height, iShape.width, iShape.channels, iShape.count);
	printf("args.strideH: %i, args.strideW: %i, args.padH: %i, args.padW: %i \n", args.strideH, args.strideW, args.padH, args.padW);
	printf("filter.height: %i, filter.width: %i, filter.channel: %i, filter.batch: %i \n", fShape.height, fShape.width, fShape.channels, fShape.count); 

	oShape.height 	= (iShape.height + 2 * args.padH - fShape.height) / args.strideH + 1;
	oShape.width	= (iShape.width  + 2 * args.padW - fShape.width)  / args.strideW + 1;
	oShape.channels	= (fShape.count);
	oShape.count 	= 1;				//	Might scale to batch size

	printf("oShape.height: %i, oShape.width: %i, oShape.channel: %i \n", oShape.height, oShape.width, oShape.channels);

	float * in = nullptr;
	float * filter = nullptr;
	float * bias = nullptr; 
	float * out = nullptr;
	float * out_cpu = nullptr;

	int retVal;
	retVal = makeTensor(&in, iShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n" ;
		return -1;
	}
	retVal = makeTensor(&filter, fShape);
	if (retVal != 0) {
		std::cout << "Unable to make tensor \n" ;
		return -1;
	}
	retVal = makeVector(&bias, oShape.channels);
	if (retVal != 0) {
		std::cout << "Unable to make vector \n" ;
		return -1;
	}


	if(args.padH != 0 || args.padW != 0){

		TensorShape_t padShape;

		padShape.count = iShape.count;
		padShape.channels = iShape.channels;
		padShape.height = iShape.height + 2 * args.padH;
		padShape.width = iShape.width + 2 * args.padW;

		float* paddedin = (float*) malloc(tensorSize(padShape) *  sizeof(float)); 
		
		for(int ch =0; ch< padShape.channels; ch++){
			for( int i = 0; i < padShape.height; i++ ) {
				for( int j = 0; j < padShape.width; j++ ) {
					int paddedPixelPos = ch * padShape.height * padShape.width + i * padShape.width + j;

					if( i >= args.padH && i < iShape.height + args.padH &&
						j >= args.padW && j < iShape.width + args.padW ) {
						int pixelPos = ch * iShape.height * iShape.width + ( i - args.padH ) * iShape.width + ( j - args.padW);
						paddedin[paddedPixelPos] = in[pixelPos];
					} else {
						paddedin[paddedPixelPos] = 0.0;
					}
				}
			}
		}
		
		free(in);

		in = paddedin;
		iShape.width = padShape.width;
		iShape.height = padShape.height;

	}	


	if(hipMalloc(&input_d,  tensorSize(iShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(iShape) * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! INPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	if(hipMalloc(&filter_d, tensorSize(fShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(fShape) * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! FILTER MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	if(hipMalloc(&bias_d,  oShape.channels * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< oShape.channels * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! BIAS MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	std::cout << "OutShape : " << oShape << " \n";
	out = (float *) malloc (tensorSize(oShape) * sizeof(float));

	if(hipMalloc(&output_d,tensorSize(oShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(oShape) * sizeof(float);
		std::cout<< "ERROR ERROR!!!!! RUN FOR THE HILLS!!!!!OUTPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	std::cout << "Input" << "\n"; 

	for(uint32_t ch = 0; ch < iShape.channels; ch++){
		std::cout<< "Channel: "<< ch << "\n";
		for (uint32_t i = 0; i < iShape.height; i++){
			for(uint32_t j = 0; j < iShape.width; j++){
				std::cout << in[ch * iShape.width * iShape.height + i * iShape.width + j] << " ";
			}
			std::cout << "\n";
		}
		std::cout << "\n";
	}

	std::cout << "\n"; 	

	std::cout << "Filter" << "\n"; 

	for(uint32_t ch = 0; ch < fShape.channels; ch++){
		std::cout<< "Channel: "<< ch << "\n";

		for (uint32_t i = 0; i < fShape.height; i++){
			for(uint32_t j = 0; j < fShape.width; j++){
				// std::cout << output[ch * oShape.width * oShape.height + i * oShape.width + j] << " @ (" << i << ", " << j << ")" << "\n";
				std::cout << filter[ch * fShape.width * fShape.height + i * fShape.width + j] << " ";
			}
			std::cout << "\n";
		}
		std::cout << "\n";
	}

	std::cout << "Bias" << "\n";  

	for(uint32_t ch = 0; ch < oShape.channels; ch++ ){
		std::cout<< "Channel: "<< ch << "\n";
		std::cout << bias[ch] << " ";
	}

	std::cout <<"\n";

	hipMemcpy(input_d, in, tensorSize(iShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(filter_d, filter, tensorSize(fShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bias_d, bias, oShape.channels * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, oShape.channels);
    dim3 dimGrid(ceil((float)oShape.width / (float)dimBlock.x), ceil((float)oShape.height / (float)dimBlock.y));

	convLayer_gpu<<<dimGrid, dimBlock>>>(input_d, iShape, filter_d, fShape, bias_d, output_d, oShape, args, 1);

	hipMemcpy(out, output_d, tensorSize(oShape) * sizeof(float), hipMemcpyDeviceToHost);

	std::cout << "\n"; 	
	std::cout << "Output GPU" << "\n"; 

	for(uint32_t ch = 0; ch < oShape.channels; ch++){
		std::cout<< "Channel: "<< ch << "\n";
		for (uint32_t i = 0; i < oShape.height; i++){
			for(uint32_t j = 0; j < oShape.width; j++){
				std::cout << out[ch * oShape.width * oShape.height + i * oShape.width + j] << " ";
			}
			std::cout << "\n";
		}
		std::cout << "\n";
	}

	#ifndef CONV_CHECK_DISABLE
		//	STUDENT: Verify number of errors in output matrix generated by convLayer_gpu
		//	STUDENT: Compare results with CPU output
		//	STUDENT: Return error count
		out_cpu = (float *) malloc (tensorSize(oShape) * sizeof(float));
		
		auto tStart = std::chrono::high_resolution_clock::now();		

		convLayer_cpu(in, iShape, filter, fShape, bias, out_cpu, oShape, args, 1);
		
		auto tEnd= std::chrono::high_resolution_clock::now();

		std::chrono::duration<double> time_span = (tEnd- tStart);

		for(uint32_t ch = 0; ch < oShape.channels; ch++){
			for (uint32_t i = 0; i < oShape.height; i++){
				for(uint32_t j = 0; j < oShape.width; j++){
					float output_gpu = out[ch * oShape.width * oShape.height + i * oShape.width + j];
					float output_cpu = out_cpu[ch * oShape.width * oShape.height + i * oShape.width + j];
					if(floor(fabs(output_gpu - output_cpu)) != 0){
						printf("Error at (%i, %i, %i) -> Actual Value: %f GPU Value: %f\n", i, j, ch, out_cpu[ch * oShape.width * oShape.height + i * oShape.width + j], out[ch * oShape.width * oShape.height + i * oShape.width + j]);
						printf("Error at (%i, %i, %i) -> Difference: %f\n", i, j, ch, floor(output_gpu - output_cpu));
						errorCount += 1;
					}
				}
			}
		}

		std::cout << "\n"; 	
		std::cout << "Output CPU" << "\n"; 

		for(uint32_t ch = 0; ch < oShape.channels; ch++){
			std::cout<< "Channel: "<< ch << "\n";
			for (uint32_t i = 0; i < oShape.height; i++){
				for(uint32_t j = 0; j < oShape.width; j++){
					std::cout << out_cpu[ch * oShape.width * oShape.height + i * oShape.width + j] << " ";
				}
				std::cout << "\n";
			}
			std::cout << "\n";
		}	

		std::cout << "\n"; 
		std::cout << "It took " << time_span.count() << " seconds on CPU.";	

	#endif

	free(in);
	free(filter);
	free(bias);
	free(out);

	hipFree(input_d);
	hipFree(filter_d);
	hipFree(bias_d);
	hipFree(output_d);

	return errorCount;
}

__global__
void convLayer_gpu ( float * input, TensorShape iShape, 
	float * filter, TensorShape fShape, 
	float * bias, float * output, TensorShape oShape, //removed & after TensorShap 
	ConvLayerArgs args, uint32_t batchSize) {
	
	int row_gl = blockDim.y * blockIdx.y + threadIdx.y;
	int col_gl = blockDim.x * blockIdx.x + threadIdx.x;
	int channel_gl = blockDim.z * blockIdx.z + threadIdx.z;

	float conv_op = 0;

	if (col_gl < oShape.width && row_gl < oShape.height && channel_gl < oShape.channels) {

		for (uint32_t n = 0; n < batchSize; ++ n) {
			//	For each output fmap value
			//	STUDENT: Set output fmap to bias
			// O[n][m][x][y] = B[m];

			conv_op = bias[channel_gl];
			
			// output[n * oShape.channels * oShape.height * oShape.width + channel_gl * oShape.height * oShape.width + row_gl * oShape.width + col_gl] = bias[channel_gl];\
			
			for (uint32_t i = 0; i < fShape.height; ++ i) {
				for (uint32_t j = 0; j < fShape.width; ++ j) {
					for (uint32_t k = 0; k < fShape.channels; ++ k) {

						//	STUDENT: Calculate
						//	O[n][m][x][y] += 
						//		I[n][k][args.strideH * x][args.strideW * y] *
						//		W[m][k][i][j];

						uint32_t input_row = (args.strideH * row_gl) + i;
						uint32_t input_col = (args.strideW * col_gl) + j;

						float input_element = input[n * iShape.channels * iShape.height * iShape.width + k * iShape.height * iShape.width + input_row * iShape.width + input_col];
						// printf("input[%i][%i][%i][%i] is %f \n", n, k, input_row, input_col, input_element);
						float filter_element = filter[channel_gl * fShape.channels * fShape.height * fShape.width + k * fShape.height * fShape.width + i * fShape.width + j];
						// output[n * oShape.channels * oShape.height * oShape.width + channel_gl * oShape.height * oShape.width + row_gl * oShape.width + col_gl] += input_element * filter_element;
						conv_op += input_element * filter_element;
					}
				}
			}

			output[n * oShape.channels * oShape.height * oShape.width + channel_gl * oShape.height * oShape.width + row_gl * oShape.width + col_gl] = conv_op;
		
		//	STUDENT: Check by disabling activation
		//	STUDENT: Apply Activation here
			if (args.activation) {
				//	O[n][m][x][y] = ReLU( O[n][m][x][y] );
				if (output[n * oShape.channels * oShape.height * oShape.width + channel_gl * oShape.height * oShape.width + row_gl * oShape.width + col_gl] < 0){
					output[n * oShape.channels * oShape.height * oShape.width + channel_gl * oShape.height * oShape.width + row_gl * oShape.width + col_gl] = 0;
				}
			}
		}
	}
	return;
}


int runGpuGemm (int argc, char ** argv) {

	TensorShape aShape = {1, 1, 6, 4};
	TensorShape bShape = {1, 1, 4, 8};
	TensorShape cShape;
	GemmLayerArgs args = {2, 2, 1};

	evaluateGpuGemm(aShape, bShape, cShape, args);
	return 0;
}

int evaluateGpuGemm(TensorShape aShape, TensorShape bShape, 
	TensorShape & cShape, GemmLayerArgs args) {

	int errorCount = 0;
	
	float * d_a, * d_b, *d_c;

	if (aShape.width != bShape.height || aShape.channels != bShape.channels 
		|| aShape.count != bShape.count) {
		std::cout << "Dimensions dont match : " << aShape << " x " << bShape << " \n";
		return -1;
	}

	cShape.height = aShape.height;
	cShape.width = bShape.width;
	cShape.channels = aShape.channels;
	cShape.count = aShape.count;

	printf("cShape.height: %i, cShape.width: %i \n", cShape.height, cShape.width);
	

	float * a = nullptr;
	float * b = nullptr;

	makeTensor(& a, aShape);
	makeTensor(& b, bShape);

	float * c = (float *) malloc(tensorSize(cShape) * sizeof(float));
	
	if(hipMalloc(&d_a,  tensorSize(aShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(aShape) * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! INPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	if(hipMalloc(&d_b,  tensorSize(bShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(bShape) * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! INPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	if(hipMalloc(&d_c,  tensorSize(cShape) * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< tensorSize(cShape) * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! INPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	std::cout << "\n";
	std::cout << "Input a" <<"\n";	

	for (uint32_t i = 0; i < aShape.height; i++){
		for(uint32_t j = 0; j < aShape.width; j++){
			std::cout << a[i * aShape.width + j] << " ";
		}
		std::cout << "\n";
	}

	std::cout << "\n";
	std::cout << "Input b" <<"\n";

	for (uint32_t i = 0; i < bShape.height; i++){
		for(uint32_t j = 0; j < bShape.width; j++){
			std::cout << b[i * bShape.width + j] << " ";
		}
		std::cout << "\n";
	}

	hipMemcpy(d_a, a, tensorSize(aShape) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, tensorSize(bShape) * sizeof(float), hipMemcpyHostToDevice);

	int sharedMemorySize = args.tileW * args.tileH * sizeof(float); 

    dim3 dimBlock(args.tileW, args.tileH);
	dim3 dimGrid(((cShape.width + args.tileW - 1)/args.tileW), ((cShape.height + args.tileH - 1)/args.tileH));

	gemmLayer_gpu<<<dimGrid, dimBlock>>>(d_a, aShape, d_b, bShape, d_c, cShape, args, 1);

	hipMemcpy(c, d_c, tensorSize(cShape) * sizeof(float), hipMemcpyDeviceToHost);

	std::cout << "\n"; 	
	std::cout << "Output GPU" << "\n"; 

	for (uint32_t i = 0; i < cShape.height; i++){
		for(uint32_t j = 0; j < cShape.width; j++){
			std::cout << c[i * cShape.width + j] << " ";
		}
		std::cout << "\n";
	}

	#ifndef CONV_CHECK_DISABLE
		//	STUDENT: Verify number of errors in output matrix generated by convLayer_gpu
		//	STUDENT: Compare results with CPU output
		//	STUDENT: Return error count
		float* out_cpu = (float *) malloc (tensorSize(cShape) * sizeof(float));
		
		auto tStart = std::chrono::high_resolution_clock::now();		

		gemmLayer_cpu (a, aShape, b, bShape, out_cpu, cShape, args, 1);

		auto tEnd= std::chrono::high_resolution_clock::now();

		std::chrono::duration<double> time_span = (tEnd- tStart);

		for (uint32_t i = 0; i < cShape.height; i++){
			for(uint32_t j = 0; j < cShape.width; j++){
				float output_gpu = c[i * cShape.width + j];
				float output_cpu = out_cpu[i * cShape.width + j];
				if(floor(fabs(output_gpu - output_cpu)) != 0){
					printf("Error at (%i, %i) -> Actual Value: %f GPU Value: %f\n", i, j, out_cpu[i * cShape.width + j], c[i * cShape.width + j]);
					printf("Error at (%i, %i) -> Difference: %f\n", i, j, floor(output_gpu - output_cpu));
					errorCount += 1;
				}
			}
		}

		std::cout << "\n"; 	
		std::cout << "Output CPU" << "\n"; 

		for (uint32_t i = 0; i < cShape.height; i++){
			for(uint32_t j = 0; j < cShape.width; j++){
				std::cout << out_cpu[i * cShape.width + j] << " ";
			}
			std::cout << "\n";
		}

		std::cout << "\n"; 
		std::cout << "It took " << time_span.count() << " seconds on CPU.";	

	#endif

	std::cout << "\nFound " << errorCount << " / " << tensorSize(cShape) << " errors \n";

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

__global__
void gemmLayer_gpu (float * a, TensorShape aShape,
	float * b, TensorShape bShape,
	float * c, TensorShape cShape,
	GemmLayerArgs args, uint32_t batchSize) {

	int row_l = threadIdx.y;
	int col_l = threadIdx.x;

	int offsetH = blockIdx.y * args.tileH;
	int offsetW = blockIdx.x * args.tileW;

	int row_gl = blockIdx.y * blockDim.y + threadIdx.y;
	int col_gl = blockIdx.x * blockDim.x + threadIdx.x;

    const uint32_t subTilesAlongK = (aShape.width + args.tileH - 1) / args.tileH;	

	int rowIdx, colIdx;
	int subTile, subTileK, k;

	#ifdef PRINT_DEBUG
		printf("%d @ (%03d, %03d)  = %d\n", threadIdx, 
		row + offsetH, col + offsetW, IDX2R(row + offsetH, col + offsetW, TILE_W));
	#endif

     for (subTile = 0; subTile < subTilesAlongK; ++ subTile) { // Which tile on the Cshape am I on?
	
		rowIdx = row_gl; 
		colIdx = col_gl;

		//  Check bounds of actual output matrix
		if (rowIdx < cShape.height && colIdx < cShape.width) {
			if (subTile == 0)
                c[IDX2R(rowIdx, colIdx, cShape.width)] = 0; //cannot understand this

			for (subTileK = 0; subTileK < args.tileH; ++ subTileK) {
				k = subTile * args.tileH + subTileK;
				if (k < aShape.width) {
					c[IDX2R(rowIdx, colIdx, cShape.width)] += a[IDX2R(rowIdx, k, aShape.width)] * b[IDX2R(k, colIdx, bShape.width)];
				}
			}
		}
	}
}
